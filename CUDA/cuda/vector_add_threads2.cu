#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define ERR 1e-6

void verify(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        assert(fabs(out[i] - a[i] - b[i]) < ERR);
    }
    printf("OK\n");
}
__device__ void task(int n){
	long x = 0;
	for(int i = 0; i < n; i++)
		x = i+1;
	n = x;
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
	int idx = threadIdx.x;
	int bdim = blockDim.x; 
	//printf("id=%d, bdim=%d\n", idx, bdim);

    for(int i = idx; i < n; i += bdim){
        out[i] = a[i] + b[i];
		task(n);
    }
}

int main(int argc, char *argv[]){
	if(argc<2) return 0;
	long N = atol(argv[1]);
    float *a, *b, *out; float *d_a, *d_b, *d_out;
    
	a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
	
	hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    for(int i = 0; i < N; i++){ a[i] = i; b[i] = i; }

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
	
	vector_add<<<1,256>>>(d_out, d_a, d_b, N);
	hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
	
	verify(out, a, b, N);

	hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
	
	free(a);
    free(b);
    free(out);
}

