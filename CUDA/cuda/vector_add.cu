/* 
 * https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial01/
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define MAX_ERR 1e-6

void verify(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("OK\n");
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(int argc, char *argv[]){
	if(argc<2) return 0;
	long N = atol(argv[1]);
    float *a, *b, *out; float *d_a, *d_b, *d_out;
    
	a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
	
	hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    for(int i = 0; i < N; i++){ a[i] = i; b[i] = i; }

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
	
	vector_add<<<1,1>>>(d_out, d_a, d_b, N);
	hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
	
	//for(int i = 0; i < N; i++){
    //    printf("%.2f ", out[i]);
    //} printf("\n");
	
	verify(out, a, b, N);

	hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
	free(a);
    free(b);
    free(out);
}

