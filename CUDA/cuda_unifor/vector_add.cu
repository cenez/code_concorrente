#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define ERR 1e-6

void verify(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        assert(fabs(out[i] - a[i] - b[i]) < ERR);
    }
    printf("OK\n");
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(int argc, char *argv[]){
	if(argc<2) return 0;
	long N = atol(argv[1]);
    float *a, *b, *out; float *d_a, *d_b, *d_out;
    
	a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
	
	hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    for(int i = 0; i < N; i++){ a[i] = i; b[i] = i; }

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
	
	vector_add<<<1,1>>>(d_out, d_a, d_b, N);
	hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
	
	verify(out, a, b, N);

	hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
	
	free(a);
    free(b);
    free(out);
}

